#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 512
#define NUM_BLOCKS 512
/* used for debugging */
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
  if(code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s, %s, %d\n", hipGetErrorString(code), file, line);
    if(abort) exit(code);
  }
}

/* function declarations */
unsigned int getmax(unsigned int *, unsigned int);
__global__ void getmaxcu(unsigned int *, unsigned int *, unsigned int);

int main(int argc, char *argv[])
{
    unsigned int size = 0;  // The size of the array
    unsigned int i;  // loop index
    unsigned int * numbers; //pointer to the array
    
    if(argc !=2)
    {
       printf("usage: maxseq num\n");
       printf("num = size of the array\n");
       exit(1);
    }
   
    size = atol(argv[1]);

    numbers = (unsigned int *)malloc(size * sizeof(unsigned int));
    if( !numbers )
    {
       printf("Unable to allocate mem for an array of size %u\n", size);
       exit(1);
    }    

    srand(time(NULL)); // setting a seed for the random number generator
    // Fill-up the array with random numbers from 0 to size-1 
    for( i = 0; i < size; i++)
       numbers[i] = rand()  % size;

    /* define the number of blocks, a host array for the maxes, and device arrays */
    unsigned int *maxes = (unsigned int *)malloc(NUM_BLOCKS * sizeof(unsigned int));
    unsigned int *dev_num, *dev_maxes; 

    /*allocate space on the device */
    gpuErrchk(hipMalloc((void**)&dev_num, size * sizeof(unsigned int)));
    gpuErrchk(hipMalloc((void**)&dev_maxes, NUM_BLOCKS * sizeof(unsigned int)));

    /*do our copies and execute the kernel */
    gpuErrchk(hipMemcpy(dev_num, numbers, size * sizeof(unsigned int), hipMemcpyHostToDevice));
    getmaxcu<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(dev_num, dev_maxes, size);
    gpuErrchk(hipPeekAtLastError()); //debug info
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(maxes, dev_maxes, NUM_BLOCKS * sizeof(unsigned int), hipMemcpyDeviceToHost));

    /* free space on the device */
    hipFree(dev_num);
    hipFree(dev_maxes);

    /* the final max calculation is done on the host
     * at this point we have few enough values that using the gpu is not necessary */
    unsigned int overall_max = 0;
    for(i = 0; i < NUM_BLOCKS; ++i) {
      if(overall_max < maxes[i])
        overall_max = maxes[i];
    }
    
    printf(" The maximum number in the array is: %u\n", overall_max);

    free(numbers);
    free(maxes);
    exit(0);
}


/*
   input: pointer to an array of long int
          number of elements in the array
   output: the maximum number of the array
*/
unsigned int getmax(unsigned int num[], unsigned int size)
{
  unsigned int i;
  unsigned int max = num[0];

  for(i = 1; i < size; i++)
	if(num[i] > max)
	   max = num[i];

  return( max );

}

__global__ void getmaxcu(unsigned int * g_idata, unsigned int * g_odata, unsigned int size) {
  unsigned int tid = threadIdx.x;
  unsigned int bid = blockIdx.x;
  unsigned int i = bid * blockDim.x + tid;

  /* find the maximum value of each block using a reduction */
  if(i < size) {
    unsigned int stride;
    for(stride = THREADS_PER_BLOCK / 2; stride > 0; stride >>= 1) {
      if(tid < stride) {
        if(g_idata[tid] < g_idata[tid + stride])
          g_idata[tid] = g_idata[tid + stride];
      }
    }
  }
  __syncthreads();

  /*write the result of each block to the output array */
  if(tid == 0)
    g_odata[bid] = g_idata[0];
}